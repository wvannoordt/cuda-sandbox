
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define DEBUG false


// Simple SAXPY operation.
__global__ void cuda_saxpy_mat_1(int N, float a, float *x, float *y)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < N*N)
	{
		y[index] = a * x[index] + y[index];
	}
}

void wait()
{
	puts("Press any key to continue...");
	getchar();
}

// Heavy lifting
double run_computation(bool on_gpu, int N)
{
#if(DEBUG)
	printf("N = %d\n\n", N);
#endif

	float a = 6.0f;
	float *host_x, *host_y, *device_x, *device_y;
	
	host_x = (float*)malloc(N*N*sizeof(float));
	host_y = (float*)malloc(N*N*sizeof(float));

	if (on_gpu)
	{
		hipMalloc(&device_x, N*N*sizeof(float)); 
		hipMalloc(&device_y, N*N*sizeof(float));
	}
	dim3 blocks_per_grid((N*N+255)/256, 1, 1);
	dim3 threads_per_block(256, 1, 1);
	for (int j = 0; j < N; j++)
	{
		for (int i = 0; i < N; i++) 
		{
			*(host_x + i*N + j) = 1.0f;
			*(host_y + i*N + j) = 3.0f;
		}
	}

#if (DEBUG)
	printf("Initializing...\n");
	printf("x[0][0]     = %f\n", *(host_x + 0*N + 0));
	printf("...\n");
	printf("x[N-1][N-1] = %f\n\n", *(host_x + (N-1)*N + (N-1)));

	printf("y[0][0]     = %f\n", *(host_y + 0*N + 0));
	printf("...\n");
	printf("y[N-1][N-1] = %f\n\n", *(host_y + (N-1)*N + (N-1)));
#endif

	if (on_gpu)
	{
		hipMemcpy(device_x, host_x, N*N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(device_y, host_y, N*N*sizeof(float), hipMemcpyHostToDevice);
	}
	
#if (DEBUG)
	printf("Computing y[i][j] = 6*x[i][j] + y[i][j]\n\n");
#endif

	clock_t start, end;
	double time_used;
	if (on_gpu)
	{
		// Runs on the GPU.
		start = clock();
		cuda_saxpy_mat_1<<<blocks_per_grid,threads_per_block>>>(N, a, device_x, device_y);
		
		// This is only a fair comparison if the device synchronization is invoked, since it awaits the end of the computation.
		hipDeviceSynchronize();
		end = clock();
	}
	else
	{
		// Runs on the CPU.
		start = clock();
		for (int j = 0; j < N; j++)
		{
			for (int i = 0; i < N; i++) 
			{
				*(host_y + i*N + j) = a * *(host_x + i*N + j) + *(host_y + i*N + j);
			}
		}
		end = clock();
	}
	
	time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	if (on_gpu)
	{
		hipMemcpy(host_y, device_y, N*N*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
	}
	
#if (DEBUG)
	printf("Outputting...\n");
	printf("x[0][0]     = %f\n", *(host_x + 0*N + 0));
	printf("...\n");
	printf("x[N-1][N-1] = %f\n\n", *(host_x + (N-1)*N + 0*(N-1)));

	printf("y[0][0]     = %f\n", *(host_y + 0*N + 0));
	printf("...\n");
	printf("y[N-1][N-1] = %f\n\n", *(host_y + (N-1)*N + (N-1)));
	
	printf("Elapsed time: %f ms\n\n", 1e3*time_used);
#endif
	if (on_gpu)
	{
		hipFree(device_x);
		hipFree(device_y);
	}
#if (DEBUG)
	if (on_gpu)
	{
		printf("Look how fast that was!\n\n\n");
	}
	else
	{
		printf("Look how fast that wasn't!\n\n\n");
	}
#endif
	if (*(host_y + (N-1)*N + (N-1)) < 5.0f)
	{
		printf("Error detected. Stop.\n\n");
		exit(1);
	}
	free(host_x);
	free(host_y);
	return 1e3*time_used;
}

int main() 
{
	// Benchmarks a SAXPY matrix operation on CPU vs GPU
	
	// Define the dimensions of the matrix. GPU performance overtakes CPU performance around N = 2^11 - 2^12 (64 MB).
	// Program crashes for N = 2^15 (4 GB).
	int N = 1<<14;
	int N2 = N*N;
	
	// Output
	printf("Runtimes for N*N = %d (%f MB):\n\n", N2, sizeof(float)*N2/(1024.0f*1024.0f));
	
	clock_t start, end;
	double time_used_total_cpu, time_used_total_gpu;
	
	
	// Run on GPU first
	start = clock();
	double gpu_time = run_computation(true, N);
	end = clock();
	time_used_total_gpu = 1e3*((double) (end - start)) / CLOCKS_PER_SEC;
	
	// Run on CPU
	start = clock();
	double cpu_time = run_computation(false, N);
	end = clock();
	time_used_total_cpu = 1e3*((double) (end - start)) / CLOCKS_PER_SEC;
	
	// Output
	printf("    CPU computation: %f ms\n", cpu_time);
	printf("    CPU total:       %f ms\n\n", time_used_total_cpu);
	printf("    GPU computation: %f ms\n", gpu_time);
	printf("    GPU total:       %f ms\n\n", time_used_total_gpu);
	return 0;
}
